
#include <hip/hip_runtime.h>
extern "C"
__global__ void computeColorDif(int length, int *rgb_ref, int *rgb_poly, double *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int color1 = rgb_ref[i];
    int color2 = rgb_poly[i];
    int color1b = color1 & 0xff;
    int color1g = (color1 & 0xff00) >> 8;
    int color1r = (color1 & 0xff0000) >> 16;
    int color1a = (color1 & 0xff000000) >> 24;
    int color2b = color2 & 0xff;
    int color2g = (color2 & 0xff00) >> 8;
    int color2r = (color2 & 0xff0000) >> 16;
    int color2a = (color2 & 0xff000000) >> 24;
    double dr = (double)color1r - color2r;
    double dg = (double)color1g - color2g;
    double db = (double)color1b - color2b;
    double da = (double)color1a - color2a;
    result[i] = sqrt(dr * dr + dg * dg + db * db + da * da)/length;
};